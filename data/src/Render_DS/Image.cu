#include "hip/hip_runtime.h"

#include "Image.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"

__host__ __device__
glm::vec3 unsign_vec3(unsigned int v) {
	glm::vec3 ret(0.0f);
	ret.r = reinterpret_cast<unsigned char*>(&(ret))[0]/255.0f;
	ret.g = reinterpret_cast<unsigned char*>(&(ret))[1]/255.0f;
	ret.b = reinterpret_cast<unsigned char*>(&(ret))[2]/255.0f;
	return ret;
}

__host__ __device__
unsigned int vec3_unsign(glm::vec3 v) {
	unsigned int ret=0xff000000;
	reinterpret_cast<unsigned char*>(&(ret))[0] = (unsigned char)(255.0 * v.r);
	reinterpret_cast<unsigned char*>(&(ret))[1] = (unsigned char)(255.0 * v.g);
	reinterpret_cast<unsigned char*>(&(ret))[2] = (unsigned char)(255.0 * v.b);
	return ret;
}

bool image::save_image(const std::string fname){
	std::vector<unsigned int> buffer(w * h);
	for(int i = 0; i < w * h; ++i) {
		buffer[i] = vec3_unsign(glm::clamp(pixels[i], 0.0f, 1.0f));
	}

	return stbi_write_png(fname.c_str(), w, h, 4, buffer.data(), w*4);
}

image image::operator+(const image &rhs) {
	image ret(w, h);

	for(int i = 0; i < w * h; ++i) {
		ret.pixels[i] = pixels[i] + rhs.pixels[i];
	}
	return ret;
}

image image::operator*(const float &rhs) {
	image ret = *this;

	for(int i = 0; i < w * h; ++i) {
		ret.pixels[i] = pixels[i] * rhs;
	}
	return ret;
}

__host__ __device__
void set_pixel(vec3 c, unsigned int& p) {
	reinterpret_cast<unsigned char*>(&p)[0] = (unsigned char)(255.0 * c.x);
	reinterpret_cast<unsigned char*>(&p)[1] = (unsigned char)(255.0 * c.y);
	reinterpret_cast<unsigned char*>(&p)[2] = (unsigned char)(255.0 * c.z);
	reinterpret_cast<unsigned char*>(&p)[3] = (unsigned char)(255);
}

__global__
void reset_pixel(const vec3 c, glm::vec3* array, int w, int h) {
	// use thread id as i, j
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int jdx = blockDim.y * blockIdx.y + threadIdx.y;

	int i_stride = blockDim.x * gridDim.x;
	int j_stride = blockDim.y * gridDim.y;

	// iterate over the output image
	for (int j = jdx; j < h; j += j_stride) 
		for (int i = idx; i < w; i += i_stride) {
			// set_pixel(pixel_value, pixels[(cur_ppc._height - 1 - j) * cur_ppc._width + i]);
            int ind = (h-1-j) * w + i;
            array[ind] = c;
		}
}

__global__
void to_unsigned_array(int w, int h, int patch_size, glm::vec3* array_a, unsigned int* array_out) {
	// use thread id as i, j
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int jdx = blockDim.y * blockIdx.y + threadIdx.y;

	int i_stride = blockDim.x * gridDim.x;
	int j_stride = blockDim.y * gridDim.y;
	float weight = 1.0f/(patch_size * patch_size);

	// iterate over the output image
	for (int j = jdx; j < h; j += j_stride) 
		for (int i = idx; i < w; i += i_stride) {
			// set_pixel(pixel_value, pixels[(cur_ppc._height - 1 - j) * cur_ppc._width + i]);
			int ind = (h-1-j) * w + i;
            set_pixel(array_a[ind] * weight, array_out[ind]);
		}
}

__global__
void add_array(int w, int h, glm::vec3* array_a, glm::vec3* array_out) {
	// use thread id as i, j
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int jdx = blockDim.y * blockIdx.y + threadIdx.y;

	int i_stride = blockDim.x * gridDim.x;
	int j_stride = blockDim.y * gridDim.y;

	// iterate over the output image
	for (int j = jdx; j < h; j += j_stride) 
		for (int i = idx; i < w; i += i_stride) {
			// set_pixel(pixel_value, pixels[(cur_ppc._height - 1 - j) * cur_ppc._width + i]);
            int ind = (h-1-j) * w + i;
            array_out[ind] += array_a[ind];
		}
}
